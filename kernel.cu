#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
//----------------------------------------------------------------------------
using namespace std;
//----------------------------------------------------------------------------
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
//----------------------------------------------------------------------------


struct eDataList {
	bool* index;
	short depth;
};

struct eData {
	eDataList* data;
	bool* isPossible;
	short current;
};

struct NonoData {
	eData* row_data;
	eData* col_data;
};

struct InputHint {
	short num;
	short* hint;
};

struct NonoInput {
	short row_size;
	short col_size;
	InputHint* row_data;
	InputHint* col_data;
};

//Find gaps
__device__ void MakeCase(short index, short num, short sum, short* g, short* gg, short *depth)
{
	if (index >= num) return;
	else if (index == num-1 ) {
		index++;
		for (int i = 0; i < sum - num; i++)
		{
			MakeCase(index, num, sum, g, gg, depth);
			g[index]++;
		}
		short temp = 0;
		for (int j = 0; j < index; j++) temp += g[j];
		if (temp <= sum)
		{
			for (int j = 0; j < num; j++) gg[depth[0]*num+j]= g[j];
			depth[0]++;			
		}
		__syncthreads();
		temp = 0;
		g[index] = 1;
	}
	else {
		index++;
		for (int i = 0; i <= sum - num + 1; i++)
		{
			MakeCase(index, num, sum, g, gg, depth);
			g[index]++;
		}
		if (index != 0 && index!=num)g[index] = 1;
	}
}

__global__ void SolveNono(NonoInput* ni, NonoData* nd,short row_size, short col_size, short* row_hints, short* col_hints, short row_num, short col_num, short* pre_row, short* pre_col, bool* results)
{
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//Transform input data to NonoData
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	short ty = threadIdx.y;
	short tx = threadIdx.x;
	
	extern __shared__ char* ds_Pout;

	//initialize element of ds_Pout as 2
	//0: blanked, 1: colored, 2: not determined
	ds_Pout = new char[row_size *(col_size)];
	ds_Pout[ty * (col_size) + tx] = 2;
	

	ni->row_size = row_size;
	ni->col_size = col_size;
	ni->row_data = new InputHint[row_size];
	ni->col_data = new InputHint[col_size];

	if(ty<(row_size)-1) ni->row_data[ty].num = pre_row[ty+1]-pre_row[ty];
	else ni->row_data[ty].num = row_num - pre_row[ty];

	ni->row_data[ty].hint = new short[ni->row_data[ty].num];
	__syncthreads();
	if(tx==0)	for(int i=0; i< ni->row_data[ty].num; i++) ni->row_data[ty].hint[i] = row_hints[pre_row[ty] + i];

	if (tx < (col_size)-1) ni->col_data[tx].num = pre_col[tx + 1] - pre_col[tx];
	else  ni->col_data[tx].num = col_num - pre_col[tx];

	ni->col_data[tx].hint = new short[ni->col_data[tx].num];
	__syncthreads();
	if (ty == 0) for (int i = 0; i < ni->col_data[tx].num; i++) ni->col_data[tx].hint[i] = col_hints[pre_col[tx] + i];
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//End of transform
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@


	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//Start to make all number of possible case
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//Initializing data structure for nonodata
	nd->row_data = new eData[row_size];
	nd->col_data = new eData[col_size];

	nd->row_data[ty].data = new eDataList[col_size];
	nd->row_data[ty].data[tx].depth = 0;

	nd->col_data[tx].data = new eDataList[row_size];
	nd->col_data[tx].data[tx].depth = 0;

	//starting to make row case
	short Hsum = 0; //sum of hint in row
	for (int i = 0; i < ni->row_data[ty].num; i++) Hsum += ni->row_data[ty].hint[i]; //calculate Hsum for row
	short* gap;
	gap = new short[(ni->row_data[ty].num)];
	
	short* depth;
	depth = new short[1];
	depth[0] = 0;
	
	gap[0] = 0;
	
	short* gg;
	gg = new short[10000];
	for (int i = 1; i < ni->row_data[ty].num; i++) gap[i] = 1;
	if (ni->row_data[ty].hint[0] == 0)//make every possible blank for row
	{
		depth[0] = 1;
		nd->row_data[ty].data[tx].depth = 1;
		nd->row_data[ty].data[tx].index = new bool[1];
		nd->row_data[ty].data[tx].index[0] = false;
	}
	else if (ni->row_data[ty].hint[0] == col_size)
	{
		depth[0] = 1;
		nd->row_data[ty].data[tx].depth = 1;
		nd->row_data[ty].data[tx].index = new bool[1];
		nd->row_data[ty].data[tx].index[0] = true;
	}
	else {
		for (int i = 0; i <= col_size - Hsum - ni->row_data[ty].num + 1; i++)
		{
			gap[0] = i;
			MakeCase(0, (ni->row_data[ty].num), col_size - Hsum, gap, gg, depth);
		}
	}
	__syncthreads();
	
	nd->row_data[ty].data[tx].depth = depth[0]; //updating depth information for each row
	nd->row_data[ty].data[tx].index = new bool[nd->row_data[ty].data[tx].depth]; //initializing index according to the depth
	nd->row_data[ty].current = depth[0];
	nd->row_data[ty].isPossible = new bool[nd->row_data[ty].data[tx].depth];

	for (int i = 0; i < nd->row_data[ty].data[tx].depth; i++) nd->row_data[ty].isPossible[i] = true;
	for (int i = 0; i < nd->row_data[ty].data[tx].depth; i++)
	{
		nd->row_data[ty].isPossible[i] = true;
		for (int j = 0; j < ni->row_data[ty].num; j++)
		{
			gap[j] = gg[i* ni->row_data[ty].num+j]; //make gap according to ith depth
		}
		
		__syncthreads();
		short temp1;
		short temp2 = 0;

		for (int k = 0; k < ni->row_data[ty].num; k++) //make row case
		{
			temp1 = temp2 + gap[k];
			temp2 = temp1 + ni->row_data[ty].hint[k];
			if (tx < temp1)
			{
				nd->row_data[ty].data[tx].index[i] = false;
				break;
			}
			else if (tx < temp2)
			{
				nd->row_data[ty].data[tx].index[i] = true;
				break;
			}
			else
			{
				if (k >= ((ni->row_data[ty].num) - 1)) {
					nd->row_data[ty].data[tx].index[i] = false;
					break;
				}
			}
		}
		__syncthreads();
	}
	__syncthreads();
	free(gg);
	free(gap);
	//short* ngap;
	gg = new short[10000];
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@end of make row case, and start to make column case
	Hsum = 0;
	gap = new short[ni->col_data[tx].num];
	//__syncthreads();

	for (int i = 0; i < ni->col_data[tx].num; i++) Hsum += ni->col_data[tx].hint[i]; //calculate Hsum for column
	for (int i = 1; i < ni->col_data[tx].num; i++) gap[i] = 1;
	depth[0] = 0;
	gap[0] = 0;
	if (ni->col_data[tx].hint[0]==0)//make every possible blank for col
	{
		//printf("hint 0 case\n");
		depth[0] = 1;
		nd->col_data[tx].data[ty].depth = 1;
		nd->col_data[tx].data[ty].index = new bool[1];
		nd->col_data[tx].data[ty].index[0] = false;
	}
	else if (ni->col_data[tx].hint[0] == ni->row_size) 
	{
		//printf("hint full case\n");
		depth[0] = 1;
		nd->col_data[tx].data[ty].depth = 1;
		nd->col_data[tx].data[ty].index = new bool[1];
		nd->col_data[tx].data[ty].index[0] = true;
	}
	else 
	{
		for (int i = 0; i <= row_size - Hsum - ni->col_data[tx].num + 1; i++)
		{
			gap[0] = i;
			MakeCase(0, (ni->col_data[tx].num), row_size - Hsum, gap, gg, depth);
		}
	}
	__syncthreads();

	nd->col_data[tx].data[ty].depth = depth[0]; //updating depth information for each column
	nd->col_data[tx].data[ty].index = new bool[nd->col_data[tx].data[ty].depth]; //initializing index according to the depth
	nd->col_data[tx].current = depth[0];
	nd->col_data[tx].isPossible = new bool[depth[0]];
	for (int i = 0; i < nd->col_data[tx].data[ty].depth; i++) nd->col_data[tx].isPossible[i] = true;
	for (int i = 0; i < nd->col_data[tx].data[ty].depth; i++)
	{
		for (int j = 0; j < ni->col_data[tx].num; j++)
		{
			gap[j] = gg[i * ni->col_data[tx].num + j]; //make gap according to ith depth
		}

		__syncthreads();
		short temp1;
		short temp2 = 0;

		for (int k = 0; k < ni->col_data[tx].num; k++) //make row case
		{
			temp1 = temp2 + gap[k];
			temp2 = temp1 + ni->col_data[tx].hint[k];
			//	__syncthreads();
			if (ty < temp1)
			{
				nd->col_data[tx].data[ty].index[i] = false;
				break;
			}
			else if (ty < temp2)
			{
				nd->col_data[tx].data[ty].index[i] = true;
				break;
			}
			else
			{
				if (k >= ((ni->col_data[tx].num) - 1)) {
					nd->col_data[tx].data[ty].index[i] = false;
					break;
				}
			}
		}
		__syncthreads();
	}
	free(gg);
	__syncthreads();

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//End of making all number of possible case
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	ds_Pout[ty * ni->col_size + tx] = 2;
	bool isContinue = true;
	int count = 1;
	short row_index;
	short col_index;
	if (nd->row_data[ty].current == 1) ds_Pout[ty * ni->col_size + tx] = nd->row_data[ty].data[tx].index[0];
	if (nd->col_data[tx].current == 1) ds_Pout[ty * ni->col_size + tx] = nd->col_data[tx].data[ty].index[0];
	__syncthreads();
	
	while (isContinue && count <200) {

		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//Find first possible index
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		for (int i = 0; i < nd->row_data[ty].data[tx].depth; i++) {
			if (nd->row_data[ty].isPossible[i]) {
				row_index = i;
				break;
			}
		}
		for (int i = 0; i < nd->col_data[tx].data[ty].depth; i++) {
			if (nd->col_data[tx].isPossible[i]) {
				col_index = i;
				break;
			}
		}
		__syncthreads();

		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//scan row
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		if (nd->row_data[ty].current > 1 && ds_Pout[ty * ni->col_size + tx] == 2) {
			bool isIdentical = true;
			for (int i = 0; i < nd->row_data[ty].data[tx].depth; i++)
			{
				if ((nd->row_data[ty].data[tx].index[row_index] != nd->row_data[ty].data[tx].index[i])&&(nd->row_data[ty].isPossible[i]))
				{
					isIdentical = false;
					break;
				}
			}
			if (isIdentical) ds_Pout[ty * ni->col_size + tx] = nd->row_data[ty].data[tx].index[row_index];
		}
		__syncthreads();
		

		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//scan column
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		if (nd->col_data[tx].current>1 && ds_Pout[ty * ni->col_size + tx] == 2)
		{
			bool isIdentical = true;
			for (int i = 0; i < nd->col_data[tx].data[ty].depth; i++)
			{
				if ((nd->col_data[tx].data[ty].index[col_index] != nd->col_data[tx].data[ty].index[i])&&(nd->col_data[tx].isPossible[i]))
				{
					isIdentical = false;
					break;
				}
			}
			if (isIdentical) ds_Pout[ty * ni->col_size + tx] = nd->col_data[tx].data[ty].index[col_index];
		}
		
		__syncthreads();


		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//reduce column
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		if ((nd->col_data[tx].current != 1)&& (ds_Pout[ty * ni->col_size + tx] != 2))
		{
			for (int j = 0; j < nd->col_data[tx].data[ty].depth; j++)
			{
				if (ds_Pout[ty * ni->col_size + tx] != nd->col_data[tx].data[ty].index[j])
				{
					nd->col_data[tx].isPossible[j] = false;
				}
			}		
		}
		__syncthreads();
		int curcount = 0;
		for (int j = 0; j < nd->col_data[tx].data[ty].depth; j++)
		{
			if (nd->col_data[tx].isPossible[j]) curcount++;
		}
		__syncthreads();
		nd->col_data[tx].current = curcount;
		__syncthreads();

		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//reduce row
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		__syncthreads();

		if ((nd->row_data[ty].current != 1)&& (ds_Pout[ty * ni->col_size + tx] != 2))
		{
			for (int j = 0; j < nd->row_data[ty].data[tx].depth; j++)
			{
				if (ds_Pout[ty * ni->col_size + tx] != nd->row_data[ty].data[tx].index[j])
				{
					nd->row_data[ty].isPossible[j] = false;
				}
			}	
		}
		__syncthreads();

		curcount = 0;
		for (int j = 0; j < nd->row_data[ty].data[tx].depth; j++)
		{
			if (nd->row_data[ty].isPossible[j]) curcount++;
		}
		__syncthreads();
		nd->row_data[ty].current = curcount;
		__syncthreads();

		count++;
		if (nd->row_data[ty].current <= 1 && nd->col_data[tx].current <= 1) isContinue = false;

		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		//Check again
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@
		if (nd->row_data[ty].current == 1)
		{
			for(int i=0; i<nd->row_data[ty].data[tx].depth;i++) 
				if (nd->row_data[ty].isPossible[i])
				{
					ds_Pout[ty * ni->col_size + tx] = nd->row_data[ty].data[tx].index[i];
					break;
				}
		}
		if (nd->col_data[tx].current == 1)
		{
			for(int i=0; i<nd->col_data[tx].data[ty].depth;i++)
				if (nd->col_data[tx].isPossible[i])
				{
					ds_Pout[ty * ni->col_size + tx] = nd->col_data[tx].data[ty].index[i];
					break;
				}
		}

		if (nd->row_data[ty].current == 0)
		{
			for (int i = 0; i < nd->row_data[ty].data[tx].depth; i++)nd->row_data[ty].isPossible[i] = true;
			nd->row_data[ty].current = nd->row_data[ty].data[tx].depth;
		}
		if (nd->col_data[tx].current == 0)
		{
			for (int i = 0; i < nd->col_data[tx].data[ty].depth; i++)nd->col_data[tx].isPossible[i] = true;
			nd->col_data[tx].current= nd->col_data[tx].data[ty].depth;
		}
	}
	

	if(ds_Pout[ty*col_size+tx]==1) results[ty * (col_size)+tx]=true;
	else results[ty * (col_size)+tx] = false;

}

//----------------------------------------------------------------------------
void writeOutput(string oName, bool* output, short col_width, short row_width)
{
	ofstream outputFile;
	outputFile.open(oName.c_str());
	if (outputFile.is_open())
	{
		for (size_t i = 0; i < row_width; i++)
		{
			for (size_t j = 0; j < col_width; j++)
			{
				if (output[i * col_width + j]) outputFile << "��";
				else outputFile << "��";
			}
			outputFile << endl;
		}
	}
}
int main(void) {
	cout << "check input file...";
	ifstream fin("./input3.txt");
	cout << endl;

	if (!fin.is_open()) cout << "File is not opend!!";
	else if(fin.is_open()) //start of data input
	{
		short row_size, col_size, * row_hints, * col_hints, row_num, col_num, * pre_row, * pre_col;
		short d_row_size, d_col_size, * d_row_hints, * d_col_hints, * d_row_num, * d_col_num, * d_pre_row, * d_pre_col;
		bool* results, * d_results;
		NonoData* nd;
		NonoInput* ni;
		short* ds_g;

		cout << "done. \n";
		cout << "open input file...";
		fin >> row_size;
		fin >> col_size;
		fin.ignore(1);

		pre_row = new short[row_size];
		pre_col = new short[col_size];
		row_num = 0;
		col_num = 0;
		short* temp_row_data;
		temp_row_data = new short[row_size * col_size / 2];
		short* temp_col_data;
		temp_col_data = new short[col_size * row_size / 2];

		for (int ri = 0; ri < row_size; ri++)
		{
			string line;
			getline(fin, line);

			stringstream ss(line); short temp;
			short num = 0;
			while (ss >> temp)
			{
				temp_row_data[row_num + num] = temp;
				num++;
			}
			pre_row[ri] = row_num;
			row_num += num;
		}
		row_hints = new short[row_num];
		for (int ri = 0; ri < row_num; ri++)	row_hints[ri] = temp_row_data[ri];


		for (int ci = 0; ci < col_size; ci++)
		{
			string line;
			getline(fin, line);

			stringstream ss(line); short temp;
			short num = 0;
			while (ss >> temp)
			{
				temp_col_data[col_num + num] = temp;
				num++;
			}
			pre_col[ci] = col_num;
			col_num += num;
		}
		col_hints = new short[col_num];
		for (int ci = 0; ci < col_num; ci++) col_hints[ci] = temp_col_data[ci];


		cout << "done. \n";
		cout << "Row size is : " << row_size << endl;
		cout << "Column size is : " << col_size << endl;
		//end of data input

		free(temp_row_data);
		free(temp_col_data);

		d_row_size = row_size;
		d_col_size = col_size;		




		results = (bool*)malloc(sizeof(bool) * row_size * col_size);


		cout << "memory allocation to device...";
		CUDA_CHECK_RETURN(hipMalloc((void**)&ni, sizeof(NonoInput)));
		CUDA_CHECK_RETURN(hipMalloc((void**)&nd, sizeof(NonoData)));
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_row_hints, sizeof(short) * row_num));
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_col_hints, sizeof(short) * col_num));
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_pre_row, sizeof(short) * row_size));
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_pre_col, sizeof(short) * col_size));
	    CUDA_CHECK_RETURN(hipMalloc((void**)&d_results, sizeof(bool) * row_size * col_size));
		cout << "done. \n";

		cout << "memory copy from host to device...";
		CUDA_CHECK_RETURN(hipMemcpy(d_row_hints, row_hints, sizeof(short) * row_num, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_col_hints, col_hints, sizeof(short) * col_num, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_pre_row, pre_row, sizeof(short) * row_size, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_pre_col, pre_col, sizeof(short) * col_size, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_results, results, sizeof(bool) * row_size * col_size, hipMemcpyHostToDevice));

		cout << "done. \n";

		cout << "kernel launching...";
		//Time kernel launch
		hipEvent_t start, stop;
		CUDA_CHECK_RETURN(hipEventCreate(&start));
		CUDA_CHECK_RETURN(hipEventCreate(&stop));
		float elapsedTime;

		CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		dim3 size = dim3(col_size, row_size, 1);
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
		// Calling the Kernel
		//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
		SolveNono << <1, size >> > (ni, nd, d_row_size, d_col_size, d_row_hints, d_col_hints, row_num, col_num, d_pre_row, d_pre_col, d_results);
		
		CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

		CUDA_CHECK_RETURN(hipEventSynchronize(stop));
		CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipEventDestroy(start));
		CUDA_CHECK_RETURN(hipEventDestroy(stop));
		cout << "done.\nElapsed kernel time: " << elapsedTime << " ms\n";
		cout << "Copying results back to host .... ";

		cout << "copying results from device to host...\n";
		//copy back output from GPU to host
		CUDA_CHECK_RETURN(hipMemcpy(results, d_results, sizeof(bool) * row_size * col_size, hipMemcpyDeviceToHost));
		cout << "done \n";

		cout << "printing output...\n";
		//print output
		for (int row = 0; row < row_size; row++)
		{
			for (int col = 0; col < col_size; col++)
			{
				if (results[row * col_size + col]) cout << "��";
				else cout << "��";
			}
			cout << endl;
		}
		cout << "done. \n";
		writeOutput("OuputNono.txt", results, col_size, row_size);
		cout << "deleting memory...";
		//free memories
		free(row_hints), free(col_hints), free(pre_row), free(pre_col) , free(results);
		hipFree(ni), hipFree(nd), hipFree(d_row_hints), hipFree(d_col_hints), hipFree(d_pre_row), hipFree(d_pre_col) , hipFree(d_results);
		cout << "done. \n";
	}
	return 0;
}
//----------------------------------------------------------------------------
